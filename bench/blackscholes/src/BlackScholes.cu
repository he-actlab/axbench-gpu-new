#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */


#include <helper_functions.h>   // helper functions for string parsing
//#include <hip/hip_runtime_api.h>        // helper functions CUDA error checking and initialization
 // Amir
#include <fstream>
 using namespace std;
// Rima
////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



///////////////////////////////////////////////////////////////////////////////
// Polynomial approximation of cumulative normal distribution function
///////////////////////////////////////////////////////////////////////////////
__device__ inline float cndGPU(float d)
{
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float
    K = 1.0f / (1.0f + 0.2316419f * fabsf(d));

    float
    cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;


    #if defined(orig_code)
        sqrtT = sqrtf(T);
        d1 = (__logf(S / X) + (R + 0.5f * V * V) * T) / (V * sqrtT);
        d2 = d1 - V * sqrtT;
    
        CNDD1 = cndGPU(d1);
        CNDD2 = cndGPU(d2);
    
        //Calculate Call and Put simultaneously
        expRT = __expf(- R * T);
    #endif
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__global__ void BlackScholesGPU(
    float *d_CallResult,
    float *d_PutResult,
    float *d_StockPrice,
    float *d_OptionStrike,
    float *d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    ////Thread index
    //const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    ////Total number of threads in execution grid
    //const int THREAD_N = blockDim.x * gridDim.x;

    const int opt = blockDim.x * blockIdx.x + threadIdx.x;

    //No matter how small is execution grid or how large OptN is,
    //exactly OptN indices will be processed with perfect memory coalescing
    //for (int opt = tid; opt < optN; opt += THREAD_N)
    if (opt < optN)
        BlackScholesBodyGPU(
            d_CallResult[opt],
            d_PutResult[opt],
            d_StockPrice[opt],
            d_OptionStrike[opt],
            d_OptionYears[opt],
            Riskfree,
            Volatility
        );
}

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;
const int  NUM_ITERATIONS = 10;


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

#pragma parrot.start("BlackScholesBodyGPU")
    
    float
    //Results calculated by CPU for reference
    *h_CallResultCPU,
    *h_PutResultCPU,
    //CPU copy of GPU results
    *h_CallResultGPU,
    *h_PutResultGPU,
    //CPU instance of input data
    *h_StockPrice,
    *h_OptionStrike,
    *h_OptionYears;

    float
    //Results calculated by GPU
    *d_CallResult,
    *d_PutResult,
    //GPU instance of input data
    *d_StockPrice,
    *d_OptionStrike,
    *d_OptionYears;


    int i;


    h_CallResultCPU = (float *)malloc(OPT_SZ);
    h_PutResultCPU  = (float *)malloc(OPT_SZ);
    h_CallResultGPU = (float *)malloc(OPT_SZ);
    h_PutResultGPU  = (float *)malloc(OPT_SZ);
    h_StockPrice    = (float *)malloc(OPT_SZ);
    h_OptionStrike  = (float *)malloc(OPT_SZ);
    h_OptionYears   = (float *)malloc(OPT_SZ);


    hipMalloc((void **)&d_CallResult,   OPT_SZ);
    hipMalloc((void **)&d_PutResult,    OPT_SZ);
    hipMalloc((void **)&d_StockPrice,   OPT_SZ);
    hipMalloc((void **)&d_OptionStrike, OPT_SZ);
    hipMalloc((void **)&d_OptionYears,  OPT_SZ);


    srand(5347);

    // Amir
    std::ifstream dataFile(argv[1]);
    int numberOptions;
    dataFile >> numberOptions;
    float stockPrice, optionStrike, optionYear;
    // Rima

    //Generate options set
    for (i = 0; i < numberOptions; i++)
    {
        h_CallResultCPU[i] = 0.0f;
        h_PutResultCPU[i]  = -1.0f;
       
        // Amir
        dataFile >> stockPrice >> optionStrike >> optionYear;
        h_StockPrice[i] = stockPrice;
        h_OptionStrike[i] = optionStrike;
        h_OptionYears[i] =  optionYear;      
        // Rima
    }

    int optionSize = numberOptions * sizeof(float);

    hipMemcpy(d_StockPrice,  h_StockPrice,   optionSize, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionStrike, h_OptionStrike,  optionSize, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionYears,  h_OptionYears,   optionSize, hipMemcpyHostToDevice);
    hipDeviceSynchronize();


    for (i = 0; i < NUM_ITERATIONS; i++)
    {
        BlackScholesGPU<<<DIV_UP(numberOptions, 128), 128/*480, 128*/>>>(
            d_CallResult,
            d_PutResult,
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            numberOptions
        );
        
    }

    hipDeviceSynchronize();
    //Read back GPU results to compare them to CPU results
    hipMemcpy(h_CallResultGPU, d_CallResult, optionSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_PutResultGPU,  d_PutResult,  optionSize, hipMemcpyDeviceToHost);

    // Amir
    ofstream callResultFile;
    callResultFile.open(argv[2]);
    for (i = 0 ; i < numberOptions; i++)
    {
        callResultFile << h_CallResultGPU[i] << std::endl;
    }
    callResultFile.close();
    // Rima

#pragma parrot.end("BlackScholesBodyGPU")

    hipFree(d_OptionYears);
    hipFree(d_OptionStrike);
    hipFree(d_StockPrice);
    hipFree(d_PutResult);
    hipFree(d_CallResult);

    //printf("...releasing CPU memory.\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResultGPU);
    free(h_CallResultGPU);
    free(h_PutResultCPU);
    free(h_CallResultCPU);

    exit(EXIT_SUCCESS);
}
